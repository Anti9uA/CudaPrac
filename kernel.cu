#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int* vec1, * vec2;  // ȣ��Ʈ�� ����
int* gpuVec, * cpuVec;  // GPU�� CPU�� ������� ���� ����

__global__ void vecAddGPU(int* VEC1, int* VEC2, int* RESULT) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    RESULT[i] = VEC1[i] + VEC2[i];
}

void vecAddCPU(int* VEC1, int* VEC2, int* RESULT, int N) {
    for (int i = 0; i < N; i++) {
        RESULT[i] = VEC1[i] * VEC2[i];
    }
}

int main(int argc, char** argv)
{
    cout << "Vector Addition Start!!" << endl;
    int n = 100000000;
    int nBytes = n * sizeof(int);
    int block_size, block_no;
    vec1 = (int*)malloc(nBytes);
    vec2 = (int*)malloc(nBytes);
    gpuVec = (int*)malloc(nBytes);
    cpuVec = (int*)malloc(nBytes);

    int* vec1_Cuda;
    int* vec2_Cuda, 
    int* gpuVec_Cuda;
    block_size = 4;
    block_no = n / block_size;
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);

    // �ε��� ����ŭ ���Ϳ� �� �Ҵ�
    for (int i = 0; i < n; i++) {
        vec1[i] = i;
        vec2[i] = i;
    }

    // hipMalloc���� �۷ι� �޸� �Ҵ� 
    cout << "Allocating to global memory..." << endl;
    hipMalloc((void**)&vec1_Cuda, n * sizeof(int));
    hipMalloc((void**)&vec2_Cuda, n * sizeof(int));
    hipMalloc((void**)&gpuVec_Cuda, n * sizeof(int));

    // hipMemcpy()�� CPU(vec1,vec2)���� GPU(vec1_Cuda,vec2_Cuda)�� �� ����
    cout << "Copying to Device..." << endl;
    hipMemcpy(vec1_Cuda, vec1, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vec2_Cuda, vec2, n * sizeof(int), hipMemcpyHostToDevice);
    
    // GPU �۾� ����!
    cout << "GPU Start!!" << endl;
    clock_t start_gpu = clock();
    cout << "GPU running..." << endl;
    vecAddGPU <<<block_no, block_size >>> (vec1_Cuda, vec2_Cuda, gpuVec_Cuda);
    hipDeviceSynchronize();
    clock_t end_gpu = clock();
    double time_gpu = (double)(end_gpu - start_gpu) / CLOCKS_PER_SEC;

    // ����̽�(GPU)�� ��������� �ٽ� ȣ��Ʈ(CPU)�� ����
    hipMemcpy(gpuVec, gpuVec_Cuda, n * sizeof(int), hipMemcpyDeviceToHost);
    cout << "GPU time >> " << time_gpu << endl;

    // CPU �۾� ����!
    cout << "\nCPU Start!!" << endl;
    clock_t start_cpu = clock();
    cout << "CPU running..." << endl;
    vecAddCPU(vec1, vec2, cpuVec, n);
    clock_t end_cpu = clock();
    double time_cpu = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    cout << "CPU time >> "<< time_cpu << endl;

    // �޸� ����
    hipFree(vec1_Cuda);
    hipFree(vec2_Cuda);
    hipFree(gpuVec_Cuda);
    return 0;
}